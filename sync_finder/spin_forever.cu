
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdint.h>
#define ALLOC_SIZE 1024000
#define hipSuccess hipSuccess 
__global__  void AddOneKernel(uint64_t * a) {
	//for (int i = threadIdx.x; i < ALLOC_SIZE; i++) {
    while(1){
    a[0] = 1;
    }	
	//}
}

int main() {
  uint64_t *x, *d_x, *d_y;
  if (hipSuccess != hipHostMalloc(&x, ALLOC_SIZE*sizeof(uint64_t), hipHostMallocDefault))
    fprintf(stderr, "Could not alloc memory\n");
  //x = (uint64_t*)malloc(ALLOC_SIZE*sizeof(uint64_t));
  if (hipSuccess != hipMalloc(&d_x, ALLOC_SIZE*sizeof(uint64_t)))
    fprintf(stderr, "ERROR\n");
  for (int i = 0; i < ALLOC_SIZE; i++) {
    x[i] = 1;
  }
  hipStream_t stream;
  hipStreamCreate(&stream);
  if (hipSuccess != hipMemcpy(d_x, x, ALLOC_SIZE*sizeof(uint64_t),hipMemcpyHostToDevice)) 
       fprintf(stderr, "MISTAKE\n");
  AddOneKernel<<<1,1,0,stream>>>(d_x);
  //cudaMemcpy(d_y, x, ALLOC_SIZE*sizeof(uint64_t),cudaMemcpyHostToDevice);
  fprintf(stderr, "Running MempcyAsync\n");
  if (hipSuccess != hipMemcpyAsync(x, d_x, ALLOC_SIZE*sizeof(uint64_t),hipMemcpyDeviceToHost, 0))
    fprintf(stderr, "MISTAKE\n");
  fprintf(stderr, "Exiting Memcpy Async\n");
  hipDeviceSynchronize();
  fprintf(stderr, "Coming from device Synchronize\n");
}
